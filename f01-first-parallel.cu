
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Refactor firstParallel so that it can run on the GPU.
 */

__global__ void firstParallel()
{
  printf("This should be running in parallel.\n");
}

int main()
{
  /*
   * Refactor this call to firstParallel to execute in parallel
   * on the GPU.
   */


  /*
   * Some code is needed below so that the CPU will wait
   * for the GPU kernels to complete before proceeding.
   */
   
   
   firstParallel<<<5, 5>>>(); /* Five threads, five blocks of a kernel. Runs 5*5=25 times! */
   hipDeviceSynchronize();
}